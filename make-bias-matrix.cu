#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <boost/program_options.hpp>

int N = 1000;
int batch_size = 32;
int output_size = 14*14;
const int blocks = 2048;

__global__ void kernel_row(float* bias_matrix,float* bias,int output_size,int batch_size){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if( output_size <= tid )return;
	float bias_value = bias[tid];
	for(int i = batch_size-1;i >= 0;i--){
		bias_matrix[i * output_size + tid] = bias_value;
	}
}
__global__ void kernel_col(float* bias_matrix,float* bias,int output_size,int batch_size){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if( batch_size <= tid )return;
	for(int i = output_size - 1;i >= 0 ;i--){
		bias_matrix[tid * output_size + i] = bias[i];
	}
}
__global__ void kernel_col_asm(float* bias_matrix,float* bias,int output_size,int batch_size){
	asm volatile("{\n\t"
#include "converted_kernel_col_asm.ptx"
		"}"
		::"l"(bias_matrix),"l"(bias),"r"(output_size),"r"(batch_size)
			);
}

__global__ void kernel_row_asm(float* bias_matrix,float* bias,int output_size,int batch_size){
	asm volatile("{\n\t"
#include "converted_kernel_row_asm.ptx"
		"}"
		::"l"(bias_matrix),"l"(bias),"r"(output_size),"r"(batch_size)
			);
}

int main(int argc,char** argv){
	boost::program_options::options_description opt("option");
	opt.add_options()
		("help,h","show help")
		("output,o",boost::program_options::value<int>()->default_value(output_size),	"matrix output size")
		("batch,b",boost::program_options::value<int>()->default_value(batch_size),	"matrix batch size")
		("calc,c",boost::program_options::value<int>()->default_value(N),	"calculation count");
	boost::program_options::variables_map vm;
	boost::program_options::store(boost::program_options::parse_command_line(argc,argv,opt),vm);
	boost::program_options::notify(vm);
	if(vm.count("help")!=0){
		std::cout<<opt<<std::endl;
		return 0;
	}
	N = vm["calc"].as<int>();
	batch_size = vm["batch"].as<int>();
	output_size = vm["output"].as<int>();

	float *d_bias,*d_bias_matrix;
	float *h_bias,*h_bias_matrix;
	hipMalloc( (void**)&d_bias, sizeof(float) * output_size);
	hipMalloc( (void**)&d_bias_matrix, sizeof(float) * output_size * batch_size);
	hipHostMalloc( (void**)&h_bias, sizeof(float) * output_size);
	hipHostMalloc( (void**)&h_bias_matrix, sizeof(float) * output_size * batch_size);
	for(int i = 0;i < output_size;i++) h_bias[i] = (i+1)/100.0f;

	/*hipMemset( d_bias_matrix, 0, sizeof(float) * output_size * batch_size);
	hipMemcpy( d_bias, h_bias, sizeof(float) * output_size , hipMemcpyHostToDevice);
	{
		auto start = std::chrono::system_clock::now();
		for(int i = 0;i < N;i++){
			for(int j = 0;j < batch_size;j++) hipMemcpy( d_bias_matrix + j * output_size, d_bias, sizeof(float) * output_size, hipMemcpyDeviceToDevice );
		}
		hipDeviceSynchronize();
		auto stop = std::chrono::system_clock::now();
		float calc_time = std::chrono::duration_cast<std::chrono::microseconds>(stop - start).count()/(float)N;
		std::cout<<"hipMemcpy copy : "<<calc_time<<" [us]"<<std::endl;
	}*/

	hipMemset( d_bias_matrix, 0, sizeof(float) * output_size * batch_size);
	hipMemcpy( d_bias, h_bias, sizeof(float) * output_size , hipMemcpyHostToDevice);

	{
		//auto start = std::chrono::system_clock::now();
		for(int i = 0;i < N;i++){
			kernel_col<<<blocks,(batch_size+blocks-1)/blocks>>>(d_bias_matrix,d_bias,output_size,batch_size);
		}
		hipDeviceSynchronize();
		//auto stop = std::chrono::system_clock::now();
		//float calc_time = std::chrono::duration_cast<std::chrono::microseconds>(stop - start).count()/(float)N;
		//std::cout<<"kernel_col copy : "<<calc_time<<" [us]"<<std::endl;
	}
	hipMemset( d_bias_matrix, 0, sizeof(float) * output_size * batch_size);
	hipMemcpy( d_bias, h_bias, sizeof(float) * output_size , hipMemcpyHostToDevice);
	{
		//auto start = std::chrono::system_clock::now();
		for(int i = 0;i < N;i++){
			kernel_row<<<blocks,(output_size+blocks)/blocks>>>(d_bias_matrix,d_bias,output_size,batch_size);
		}
		hipDeviceSynchronize();
		//auto stop = std::chrono::system_clock::now();
		//float calc_time = std::chrono::duration_cast<std::chrono::microseconds>(stop - start).count()/(float)N;
		//std::cout<<"kernel_row copy : "<<calc_time<<" [us]"<<std::endl;
	}
	hipMemset( d_bias_matrix, 0, sizeof(float) * output_size * batch_size);
	hipMemcpy( d_bias, h_bias, sizeof(float) * output_size , hipMemcpyHostToDevice);
	{
		//auto start = std::chrono::system_clock::now();
		for(int i = 0;i < N;i++){
			kernel_col_asm<<<128,(batch_size+127)/128>>>(d_bias_matrix,d_bias,output_size,batch_size);
		}
		hipDeviceSynchronize();
		//auto stop = std::chrono::system_clock::now();
		//float calc_time = std::chrono::duration_cast<std::chrono::microseconds>(stop - start).count()/(float)N;
		//std::cout<<"kernel_asm_col copy : "<<calc_time<<" [us]"<<std::endl;
	}
	hipMemset( d_bias_matrix, 0, sizeof(float) * output_size * batch_size);
	hipMemcpy( d_bias, h_bias, sizeof(float) * output_size , hipMemcpyHostToDevice);
	{
		//auto start = std::chrono::system_clock::now();
		for(int i = 0;i < N;i++){
			kernel_row_asm<<<128,(output_size+127)/128>>>(d_bias_matrix,d_bias,output_size,batch_size);
		}
		hipDeviceSynchronize();
		//auto stop = std::chrono::system_clock::now();
		//float calc_time = std::chrono::duration_cast<std::chrono::microseconds>(stop - start).count()/(float)N;
		//std::cout<<"kernel_asm_row copy : "<<calc_time<<" [us]"<<std::endl;
	}

	hipMemcpy( h_bias_matrix, d_bias_matrix, sizeof(float) * output_size * batch_size, hipMemcpyDeviceToHost);
	/*for(int j = 0;j < output_size;j++){
		for(int i = 0;i < batch_size;i++){
			printf("%.3f ",h_bias_matrix[i * output_size + j]);
		}
		printf("\n");
	}*/

	hipFree( d_bias );
	hipHostFree( h_bias );
	hipFree( d_bias_matrix );
	hipHostFree( h_bias_matrix );
}
